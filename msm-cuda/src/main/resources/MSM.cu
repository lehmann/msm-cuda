
#include <hip/hip_runtime.h>
#include <inttypes.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
//#include <time.h>
#include <sys/time.h>
#include <sys/types.h>
#include <dirent.h>
#include <unistd.h>

#define MAX(x, y) (((x) > (y)) ? (x) : (y))
#define MIN(x, y) (((x) < (y)) ? (x) : (y))

#define MAX_STR_LEN 256

struct ponto_capturado{
    int TID;
    char *clazz;
    int time;
    double lat, lon;
    int gid;
    int stopId;
};
struct trajetoria{
	ponto_capturado** pontos;
	int qntdPontos;
};

trajetoria** trajetorias;

trajetoria* readTrajFile(char*);

double* trajectoryRawer(trajetoria*);

double euclidean(double *p1, double *p2);

void msm(double* trajA, int lengthA, double* trajB, int lengthB, double* aScore, double* bScore, double* semanticsDescriptors);

double distance(double*, int, double*, int);

int main(int argc, char *argv[]) {
	int file_count = 0;
	int len;
	DIR * dirp;
	struct dirent * entry;

	dirp = opendir("./trajetorias");
	while ((entry = readdir(dirp)) != NULL) {
        len = strlen (entry->d_name);
		if (entry->d_type == DT_REG && strcmp (".traj", &(entry->d_name[len - 5])) == 0) { /* If the entry is a regular file */
			 file_count++;
		}
	}
	closedir(dirp);
	trajetorias  = (trajetoria**) malloc(file_count*sizeof(trajetoria*));
    DIR* FD;
    struct dirent* in_file;
    if (NULL == (FD = opendir ("./trajetorias"))) {
        fprintf(stderr, "Error : Failed to open input directory\n");
        return 1;
    }
	int fileCounter = 0;
    while ((in_file = readdir(FD))) {
        len = strlen (in_file->d_name);
		if (len > 4 && in_file->d_type == DT_REG && strcmp (".traj", &(in_file->d_name[len - 5])) == 0) {
			if (!strcmp (in_file->d_name, "."))
				continue;
			if (!strcmp (in_file->d_name, ".."))    
				continue;
			char filePath[1024];
			sprintf( filePath, "%s/%s", "./trajetorias", in_file->d_name );
			trajetorias[fileCounter++] = readTrajFile(filePath);
		}
	}
	printf("Qntd arquivos lidos %d\n", file_count);
	
	double** allDistances = (double**) malloc(file_count*sizeof(double*));
	double** rawTrajs = (double**) malloc(file_count*sizeof(double*));
	for(int k = 0;k<file_count;k++) {
		rawTrajs[k] = trajectoryRawer(trajetorias[k]);
	}
	for(int k = 0;k<file_count;k++) {
		allDistances[k] = (double*) malloc(file_count*sizeof(double));
	}
	printf("Trajetorias transformadas %d\n", file_count);
	for(int k = 0;k<file_count;k++) {
		allDistances[k][k] = 0.0;
		for(int l = 0;l<file_count;l++) {
	//printf("Distance lengthA=%d, lengthB=%d\n", trajetorias[k]->qntdPontos, trajetorias[l]->qntdPontos);
			if(k<l) {
				double *trajA = rawTrajs[k];
				double *trajB = rawTrajs[l];
				double similarity = distance(trajA, trajetorias[k]->qntdPontos, trajB, trajetorias[l]->qntdPontos);
				allDistances[k][l] = similarity;
				allDistances[l][k] = similarity;
				//printf("Similaridade das trajet�rias: %.2f\n", similarity);
			}
		}
	}

	for(int i = 0; i < file_count;i++) {
		if(trajetorias[i]) {
			for(int j = 0; j < trajetorias[i]->qntdPontos;j++) {
				free(trajetorias[i]->pontos[j]);
			}
			free(trajetorias[i]);
		}
	}
	free(trajetorias);
	
	return 0;
}

double distance(double* trajA, int N, double* trajB, int M) {
	double* aScore = (double*)malloc( N*sizeof(double));
	double* bScore = (double*)malloc( N*M*sizeof(double));
	double* semanticsDescriptors = (double*)malloc( 2*2*sizeof(double));
	//GEO
	semanticsDescriptors[0] = 0.0;
	semanticsDescriptors[1] = 0.5;
	//TIME
	semanticsDescriptors[2] = 0.0;
	semanticsDescriptors[3] = 0.5;

	//printf("Distance lengthA=%d, lengthB=%d\n", N,M);
    msm( trajA, N, trajB, M, aScore, bScore, semanticsDescriptors );
    
	double parityAB = 0.0;
	for (int i = 0; i < N; i++) {
		parityAB += aScore[i];
	}

	double parityBA = 0.0;
	for (int i = 0; i < N; i++) {
		double maxScore = 0.0;
		for (int j = 0; j < M; j++) {
			maxScore = MAX(maxScore, bScore[i * M + j]);
		}
		parityBA += maxScore;
	}
	//printf("parityAB=%.2f, parityBA=%.2f\n", parityAB, parityBA );
	double similarity = (parityAB + parityBA) / (N + M);
	free(semanticsDescriptors);
	//printf("similarity=%.2f\n", similarity );
	free(bScore);
	free(aScore);
	aScore = NULL;
	bScore = NULL;
	semanticsDescriptors = NULL;
	
	return similarity;
}

void msm(double* trajA, int lengthA, double* trajB, int lengthB, double* aScore, double* bScore, double* semanticsDescriptors) {
	for(int i = 0; i < lengthA; i++) {
		double latGeoA = trajA[i * 4];
		double lonGeoA = trajA[i * 4 + 1];
		double startTimeA = trajA[i * 4 + 2];
		double endTimeA = trajA[i * 4 + 3];
		
		double geoThreshold = semanticsDescriptors[0];
		double timeThreshold = semanticsDescriptors[2];
	
		double geoWeight = semanticsDescriptors[1];
		double timeWeight = semanticsDescriptors[3];
	
		double maxScore = 0.0;
		for (int j = 0; j < lengthB; j++) {
			double latGeoB = trajB[j * 4];
			double lonGeoB = trajB[j * 4 + 1];
			double startTimeB = trajB[j * 4 + 2];
			double endTimeB = trajB[j * 4 + 3];
			double timeScore = 0.0;
			if(startTimeA < endTimeB && startTimeB < endTimeA ) {
			    double overlap = MIN(endTimeA, endTimeB) - MAX(startTimeA, startTimeB);
			    if(overlap > 0.0) {
	    			double duration = MAX(endTimeA, endTimeB) - MIN(startTimeA, startTimeB);
	    			double timeDistance = 1 - (overlap / duration);
	    			timeScore = (timeDistance <= timeThreshold ? 1 : 0) * timeWeight;
			    }
			}
			double geoB[] = {latGeoB, lonGeoB};
			double geoA[] = {latGeoA, lonGeoA};
			double geoScore = (euclidean(geoB, geoA) <= geoThreshold ? 1 : 0) * geoWeight;
			double sumScore = timeScore + geoScore;
			if(sumScore > maxScore) {
			    maxScore = sumScore;
			}
		    bScore[i * lengthB + j] = sumScore;
		}
		aScore[i] = maxScore;
	}
}

trajetoria* readTrajFile(char *filePath) {
    /* FileStream for the Library File */
    FILE *trajFile;

    /* allocation of the buffer for every line in the File */
    char *buf = (char*) malloc(MAX_STR_LEN);
    char *tmp; 

    /* if the space could not be allocaed, return an error */
    if (buf == NULL) {
        printf ("No memory\n");
        return NULL;
    }

    if ( ( trajFile = fopen( filePath, "r" ) ) == NULL ) //Reading a file
    {
        printf( "File could not be opened: %s.\n", filePath );
		return NULL;
    }
	int pointsCounter = 0;
    while (fgets(buf, MAX_STR_LEN - 1, trajFile) != NULL) {	
		pointsCounter++;
	}
    fclose(trajFile);
	ponto_capturado **traj = (ponto_capturado**) malloc(pointsCounter*sizeof(ponto_capturado*));
	trajetoria* trajetoria = new struct trajetoria;
	trajetoria->pontos = traj;
	trajetoria->qntdPontos = pointsCounter;

    if ( ( trajFile = fopen( filePath, "r" ) ) == NULL ) {
        printf( "File could not be opened: %s.\n", filePath );
		return NULL;
    }
    int i = 0;
    while (fgets(buf, MAX_STR_LEN - 1, trajFile) != NULL)
    {	

        if (strlen(buf)>0) {
	      if(buf[strlen (buf) - 1] == '\n')
	            buf[strlen (buf) - 1] = '\0';
		} else {
			if(buf[0] == '\n') {
				continue;
			}
		}

        tmp = strtok(buf, ";");
		
		traj[i] = new ponto_capturado();
		
        traj[i]->TID = atoi(tmp);

        tmp = strtok(NULL, ";");
		int len = strlen(tmp);
		traj[i]->clazz = (char*)malloc(len + 1);
		strcpy(traj[i]->clazz, tmp);

        tmp = strtok(NULL, ";");
        traj[i]->time = atoi(tmp);

        tmp = strtok(NULL, ";");
        traj[i]->lat = atof(tmp);

        tmp = strtok(NULL, ";");
        traj[i]->lon = atof(tmp);

        tmp = strtok(NULL, ";");
        traj[i]->gid = atoi(tmp);

        tmp = strtok(NULL, ";");

        if ((tmp != NULL) && (tmp[0] == '\0')) {
	        traj[i]->stopId = atoi(tmp);
        } else {
	        traj[i]->stopId = 0;
        }

		/*
        printf("index i= %d  ID: %d, %s, %d, %.8f, %.8f, %d, %d \n",i, traj[i]->TID ,
        					 traj[i]->clazz, traj[i]->time , 
        					 traj[i]->lat, traj[i]->lon,
        					 traj[i]->gid, traj[i]->stopId);
		*/
        i++;
    }
	//printf("Loaded %s - %d points\n", filePath, i);
    fclose(trajFile);
    return trajetoria;
}

double* trajectoryRawer(trajetoria* trajetoria) {
	int N = trajetoria->qntdPontos;
	double* trajA = (double*)malloc( 4*N*sizeof(double));
	for(int i = 0; i < N; i++) {
		trajA[i * 4] = trajetoria->pontos[i]->lat;
		trajA[i * 4 + 1] = trajetoria->pontos[i]->lon;
		trajA[i * 4 + 2] = trajetoria->pontos[i]->time;
		trajA[i * 4 + 3] = trajetoria->pontos[i]->time + 30;
	}
	return trajA;
}

double euclidean(double *p1, double *p2) {
	double distX = abs(p1[0] - p2[0]);
	double distXSquare = distX * distX;

	double distY = abs(p1[1] - p2[1]);
	double distYSquare = distY * distY;

	return sqrt(distXSquare + distYSquare);
}