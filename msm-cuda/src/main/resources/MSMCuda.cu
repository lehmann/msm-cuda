#include <inttypes.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define MAX(x, y) (((x) > (y)) ? (x) : (y))
#define MIN(x, y) (((x) < (y)) ? (x) : (y))

__device__ double euclidean(double *p1, double *p2);

__global__ void msm(double* trajA, int lengthA, double* trajB, int lengthB, double* aScore, double* bScore, double* semanticsDescriptors);

int main() {
	int N = 10;
	double* trajA = (double*)malloc( N*N*sizeof(double));
	double* trajB = (double*)malloc( N*N*sizeof(double));
	double* semanticsDescriptors = (double*)malloc( 2*2*sizeof(double));
	double* aScore = (double*)malloc( N*sizeof(double));
	double* bScore = (double*)malloc( N*sizeof(double));
	struct timeval tv;
	gettimeofday(&tv, NULL);
	
	double time_in_mill = (tv.tv_sec) * 1000 + (tv.tv_usec) / 1000 ; 
	for(int i = N - 1; i > -1; i--) {
		trajA[i * N] = i;
		trajA[i * N + 1] = i;
		trajA[i * N + 2] = time_in_mill - i;
		trajA[i * N + 3] = time_in_mill - (i-1);
		
		trajB[i * N] = i;
		trajB[i * N + 1] = i;
		trajB[i * N + 2] = time_in_mill - i;
		trajB[i * N + 3] = time_in_mill - (i-1);
	}
	//GEO
	semanticsDescriptors[0] = 0.0;
	semanticsDescriptors[1] = 0.5;
	//TIME
	semanticsDescriptors[2] = 0.0;
	semanticsDescriptors[3] = 0.5;
	
	double *d_trajA,*d_trajB, *d_aScore, *d_bScore, *d_semanticsDescriptors;
	hipMalloc( (void**) &d_trajA, N*N*sizeof(double) );
	hipMalloc( (void**) &d_trajB, N*N*sizeof(double) ); 

	hipMalloc( (void**) &d_semanticsDescriptors, 2*2*sizeof(double) );
	hipMalloc( (void**) &d_aScore, N*sizeof(double) );
	hipMalloc( (void**) &d_bScore, N*N*sizeof(double) );
	hipMemcpy( (void*) d_trajA, (void*) trajA, N*N*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy( (void*) d_trajB, (void*) trajB, N*N*sizeof(double), hipMemcpyHostToDevice); 
	hipMemcpy( (void*) d_semanticsDescriptors, (void*) semanticsDescriptors, 2*2*sizeof(double), hipMemcpyHostToDevice); 
	
	int THREADS = 128;
	int BLOCOS = (N/THREADS) + 1;
	
    struct timeval begin, end;
    gettimeofday(&begin, NULL);
    msm<<<BLOCOS, THREADS>>>( d_trajA, N, d_trajB, N, d_aScore, d_bScore, d_semanticsDescriptors );
    gettimeofday(&end, NULL);
    
	hipMemcpy( (void*) aScore, (void*) d_aScore, N*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy( (void*) bScore, (void*) d_bScore, N*N*sizeof(double), hipMemcpyDeviceToHost);
	
	hipFree(d_trajA); 
	hipFree(d_trajB); 
	hipFree(d_aScore);
	hipFree(d_bScore);
	hipFree(d_semanticsDescriptors); 
	 
	double parityAB = 0.0;
	for (int i = 0; i < N; i++) {
		parityAB += aScore[i];
	}

	double parityBA = 0.0;
	for (int i = 0; i < N; i++) {
		double maxScore = 0.0;
		for (int j = 0; j < N; j++) {
			maxScore = MAX(maxScore, bScore[i * N + j]);
		}
		parityBA += maxScore;
	}
	//printf("parityAB=%.2f, parityBA=%.2f\n", parityAB, parityBA );
	double similarity = (parityAB + parityBA) / (N + N);
	
	printf("Similaridade das trajet�rias: %.2f\n", similarity);
	
	return 0;
}

//extern "C"
__global__ void msm(double* trajA, int lengthA, double* trajB, int lengthB, double* aScore, double* bScore, double* semanticsDescriptors)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i>=lengthA) {
    	return;
    }
	double latGeoA = trajA[i * lengthA];
	double lonGeoA = trajA[i * lengthA + 1];
	double startTimeA = trajA[i * lengthA + 2];
	double endTimeA = trajA[i * lengthA + 3];
	
	double geoThreshold = semanticsDescriptors[0];
	double timeThreshold = semanticsDescriptors[2];

	double geoWeight = semanticsDescriptors[1];
	double timeWeight = semanticsDescriptors[3];

	double maxScore = 0.0;
	double maxGeoScore = 0.0;
	double maxTimeScore = 0.0;
	for (int j = 0; j < lengthB; j++) {
		double latGeoB = trajB[j * lengthB];
		double lonGeoB = trajB[j * lengthB + 1];
		double startTimeB = trajB[j * lengthB + 2];
		double endTimeB = trajB[j * lengthB + 3];
		double timeScore = 0.0;
		if(startTimeA < endTimeB && startTimeB < endTimeA ) {
		    double overlap = MIN(endTimeA, endTimeB) - MAX(startTimeA, startTimeB);
		    if(overlap > 0.0) {
    			double duration = MAX(endTimeA, endTimeB) - MIN(startTimeA, startTimeB);
    			double timeDistance = 1 - (overlap / duration);
    			timeScore = (timeDistance <= timeThreshold ? 1 : 0) * timeWeight;
		    }
		}
		double geoB[] = {latGeoB, lonGeoB};
		double geoA[] = {latGeoA, lonGeoA};
		double geoScore = (euclidean(geoB, geoA) <= geoThreshold ? 1 : 0) * geoWeight;
		double sumScore = timeScore + geoScore;
		if(sumScore > maxScore) {
		    maxScore = sumScore;
		    maxGeoScore = geoScore;
		    maxTimeScore = timeScore;
		}
	    	bScore[i * lengthA + j] = sumScore;
	}
	//printf("Thread %d, maxScore=%.2f, maxGeoScore=%.2f, maxTimeScore=%.2f\n", i, maxScore, maxGeoScore,maxTimeScore );
	aScore[i] = maxScore;
}

__device__ double euclidean(double *p1, double *p2)
{
	double distX = abs(p1[0] - p2[0]);
	double distXSquare = distX * distX;

	double distY = abs(p1[1] - p2[1]);
	double distYSquare = distY * distY;

	return sqrt(distXSquare + distYSquare);
}

