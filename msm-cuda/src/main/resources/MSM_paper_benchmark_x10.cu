
#include <hip/hip_runtime.h>
#include <inttypes.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
//#include <time.h>
#include <sys/time.h>
#include <sys/types.h>
#include <dirent.h>
#include <unistd.h>

#define MAX(x, y) (((x) > (y)) ? (x) : (y))
#define MIN(x, y) (((x) < (y)) ? (x) : (y))

#define MAX_STR_LEN 256

struct ponto_capturado{
    int TID;
    char *clazz;
    int time;
    double lat, lon;
    int gid;
    int stopId;
};
struct trajetoria{
	ponto_capturado** pontos;
	int qntdPontos;
};

double msm_sequential(double* trajA, int lengthA, double* trajB, int lengthB, double* semanticsDescriptors);

double euclidean_local(double *p1, double *p2);

trajetoria** trajetorias;

trajetoria* readTrajFile(char*);

double* trajectoryRawer(trajetoria*);

double distance_sequential(double*, int, double*, int);

int main(int argc, char *argv[]) {
	int file_count = 0;
	int len;
	DIR * dirp;
	struct dirent * entry;

	dirp = opendir("./trajetorias");
	while ((entry = readdir(dirp)) != NULL) {
        len = strlen (entry->d_name);
		if (entry->d_type == DT_REG && strcmp (".traj", &(entry->d_name[len - 5])) == 0) { /* If the entry is a regular file */
			 file_count++;
		}
	}
	closedir(dirp);
	trajetorias  = (trajetoria**) malloc(file_count*sizeof(trajetoria*));
    DIR* FD;
    struct dirent* in_file;
    if (NULL == (FD = opendir ("./trajetorias"))) {
        fprintf(stderr, "Error : Failed to open input directory\n");
        return 1;
    }
	int fileCounter = 0;
    while ((in_file = readdir(FD))) {
        len = strlen (in_file->d_name);
		if (len > 4 && in_file->d_type == DT_REG && strcmp (".traj", &(in_file->d_name[len - 5])) == 0) {
			if (!strcmp (in_file->d_name, "."))
				continue;
			if (!strcmp (in_file->d_name, ".."))    
				continue;
			char filePath[1024];
			sprintf( filePath, "%s/%s", "./trajetorias", in_file->d_name );
			trajetorias[fileCounter++] = readTrajFile(filePath);
		}
	}
	printf("Qntd arquivos lidos %d\n", file_count);
	
	double** allDistances = (double**) malloc(file_count*sizeof(double*));
	double** rawTrajs = (double**) malloc(file_count*sizeof(double*));
	for(int k = 0;k<file_count;k++) {
		rawTrajs[k] = trajectoryRawer(trajetorias[k]);
	}
	for(int k = 0;k<file_count;k++) {
		allDistances[k] = (double*) malloc(file_count*sizeof(double));
	}
	printf("Trajetorias transformadas %d\n", file_count);
	
	struct timeval begin, end;
	
	printf("Executando algoritmo original\n");
	gettimeofday(&begin, NULL);
	for(int iteration = 0; iteration < 10; iteration++) {
		for(int k = 0;k<file_count;k++) {
			allDistances[k][k] = 0.0;
			for(int l = 0;l<file_count;l++) {
				if(k<l) {
					double *trajA = rawTrajs[k];
					double *trajB = rawTrajs[l];
					double similarity = distance_sequential(trajA, trajetorias[k]->qntdPontos, trajB, trajetorias[l]->qntdPontos);
					allDistances[k][l] = similarity;
					allDistances[l][k] = similarity;
				}
			}
		}
	}
	gettimeofday(&end, NULL);
    float cpuTime = 1000000*(float)(end.tv_sec - begin.tv_sec);
    cpuTime +=  (float)(end.tv_usec - begin.tv_usec);

	printf("Tempo de execu��o para a constru��o da matriz de similaridade entre todas as trajet�rias: %9.2f microssegundos\n", cpuTime);
	for(int i = 0; i < file_count;i++) {
		if(trajetorias[i]) {
			for(int j = 0; j < trajetorias[i]->qntdPontos;j++) {
				free(trajetorias[i]->pontos[j]);
			}
			free(trajetorias[i]);
		}
	}
	free(trajetorias);
	
	return 0;
}

trajetoria* readTrajFile(char *filePath) {
    /* FileStream for the Library File */
    FILE *trajFile;

    /* allocation of the buffer for every line in the File */
    char *buf = (char*) malloc(MAX_STR_LEN);
    char *tmp; 

    /* if the space could not be allocaed, return an error */
    if (buf == NULL) {
        printf ("No memory\n");
        return NULL;
    }

    if ( ( trajFile = fopen( filePath, "r" ) ) == NULL ) //Reading a file
    {
        printf( "File could not be opened: %s.\n", filePath );
		return NULL;
    }
	int pointsCounter = 0;
    while (fgets(buf, MAX_STR_LEN - 1, trajFile) != NULL) {	
		pointsCounter++;
	}
    fclose(trajFile);
	ponto_capturado **traj = (ponto_capturado**) malloc(pointsCounter*sizeof(ponto_capturado*));
	trajetoria* trajetoria = new struct trajetoria;
	trajetoria->pontos = traj;
	trajetoria->qntdPontos = pointsCounter;

    if ( ( trajFile = fopen( filePath, "r" ) ) == NULL ) {
        printf( "File could not be opened: %s.\n", filePath );
		return NULL;
    }
    int i = 0;
    while (fgets(buf, MAX_STR_LEN - 1, trajFile) != NULL)
    {	

        if (strlen(buf)>0) {
	      if(buf[strlen (buf) - 1] == '\n')
	            buf[strlen (buf) - 1] = '\0';
		} else {
			if(buf[0] == '\n') {
				continue;
			}
		}

        tmp = strtok(buf, ";");
		
		traj[i] = new ponto_capturado();
		
        traj[i]->TID = atoi(tmp);

        tmp = strtok(NULL, ";");
		int len = strlen(tmp);
		traj[i]->clazz = (char*)malloc(len + 1);
		strcpy(traj[i]->clazz, tmp);

        tmp = strtok(NULL, ";");
        traj[i]->time = atoi(tmp);

        tmp = strtok(NULL, ";");
        traj[i]->lat = atof(tmp);

        tmp = strtok(NULL, ";");
        traj[i]->lon = atof(tmp);

        tmp = strtok(NULL, ";");
        traj[i]->gid = atoi(tmp);

        tmp = strtok(NULL, ";");

        if ((tmp != NULL) && (tmp[0] == '\0')) {
	        traj[i]->stopId = atoi(tmp);
        } else {
	        traj[i]->stopId = 0;
        }

        i++;
    }
	//printf("Loaded %s - %d points\n", filePath, i);
    fclose(trajFile);
    return trajetoria;
}

double* trajectoryRawer(trajetoria* trajetoria) {
	int N = trajetoria->qntdPontos;
	double* trajA = (double*)malloc( 4*N*sizeof(double));
	for(int i = 0; i < N; i++) {
		trajA[i * 4] = trajetoria->pontos[i]->lat;
		trajA[i * 4 + 1] = trajetoria->pontos[i]->lon;
		trajA[i * 4 + 2] = trajetoria->pontos[i]->time;
		trajA[i * 4 + 3] = trajetoria->pontos[i]->time + 30;
	}
	return trajA;
}

double distance_sequential(double* trajA, int N, double* trajB, int M) {
	double* semanticsDescriptors = (double*)malloc( 2*2*sizeof(double));
	//GEO
	semanticsDescriptors[0] = 0.0;
	semanticsDescriptors[1] = 0.5;
	//TIME
	semanticsDescriptors[2] = 0.0;
	semanticsDescriptors[3] = 0.5;

    double similarity = msm_sequential( trajA, N, trajB, M, semanticsDescriptors );
    
	free(semanticsDescriptors);
	semanticsDescriptors = NULL;
	
	return similarity;
}

double msm_sequential(double* trajA, int lengthA, double* trajB, int lengthB, double* semanticsDescriptors) {
	
	double geoThreshold = semanticsDescriptors[0];
	double timeThreshold = semanticsDescriptors[2];

	double geoWeight = semanticsDescriptors[1];
	double timeWeight = semanticsDescriptors[3];
	double parityAB = 0.0;
	double parityBA = 0.0;
	for(int i = 0; i < lengthA; i++) {
		double latGeoA = trajA[i * 4];
		double lonGeoA = trajA[i * 4 + 1];
		double startTimeA = trajA[i * 4 + 2];
		double endTimeA = trajA[i * 4 + 3];

		double maxScore = 0.0;
		for (int j = 0; j < lengthB; j++) {
			double latGeoB = trajB[j * 4];
			double lonGeoB = trajB[j * 4 + 1];
			double startTimeB = trajB[j * 4 + 2];
			double endTimeB = trajB[j * 4 + 3];
			double timeScore = 0.0;
			if(startTimeA < endTimeB && startTimeB < endTimeA ) {
			    double overlap = MIN(endTimeA, endTimeB) - MAX(startTimeA, startTimeB);
			    if(overlap > 0.0) {
	    			double duration = MAX(endTimeA, endTimeB) - MIN(startTimeA, startTimeB);
	    			double timeDistance = 1 - (overlap / duration);
	    			timeScore = (timeDistance <= timeThreshold ? 1 : 0) * timeWeight;
			    }
			}
			double geoB[] = {latGeoB, lonGeoB};
			double geoA[] = {latGeoA, lonGeoA};
			double geoScore = (euclidean_local(geoB, geoA) <= geoThreshold ? 1 : 0) * geoWeight;
			double sumScore = timeScore + geoScore;
			if(sumScore > maxScore) {
			    maxScore = sumScore;
			}
		}
		parityAB += maxScore;
	}
	for(int i = 0; i < lengthB; i++) {
		double latGeoB = trajB[i * 4];
		double lonGeoB = trajB[i * 4 + 1];
		double startTimeB = trajB[i * 4 + 2];
		double endTimeB = trajB[i * 4 + 3];

		double maxScore = 0.0;
		for (int j = 0; j < lengthA; j++) {
			double latGeoA = trajA[j * 4];
			double lonGeoA = trajA[j * 4 + 1];
			double startTimeA = trajA[j * 4 + 2];
			double endTimeA = trajA[j * 4 + 3];
			double timeScore = 0.0;
			if(startTimeA < endTimeB && startTimeB < endTimeA ) {
			    double overlap = MIN(endTimeA, endTimeB) - MAX(startTimeA, startTimeB);
			    if(overlap > 0.0) {
	    			double duration = MAX(endTimeA, endTimeB) - MIN(startTimeA, startTimeB);
	    			double timeDistance = 1 - (overlap / duration);
	    			timeScore = (timeDistance <= timeThreshold ? 1 : 0) * timeWeight;
			    }
			}
			double geoB[] = {latGeoB, lonGeoB};
			double geoA[] = {latGeoA, lonGeoA};
			double geoScore = (euclidean_local(geoB, geoA) <= geoThreshold ? 1 : 0) * geoWeight;
			double sumScore = timeScore + geoScore;
			if(sumScore > maxScore) {
			    maxScore = sumScore;
			}
		}
		parityBA += maxScore;
	}
	return (parityAB + parityBA) / (lengthA + lengthB);
}

double euclidean_local(double *p1, double *p2)
{
	double distX = abs(p1[0] - p2[0]);
	double distXSquare = distX * distX;

	double distY = abs(p1[1] - p2[1]);
	double distYSquare = distY * distY;

	return sqrt(distXSquare + distYSquare);
}
